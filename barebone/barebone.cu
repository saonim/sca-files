
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 1; 
const int blocksize = 1; 

__global__ void kernelFunc() {

}

int main() {
  int b[N] = {4};
  int *bd;
  const int isize = N*sizeof(int);
  printf("%i", *b);
  hipMalloc( (void**)&bd, isize ); 
  hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 

  // Allocate a big chunk of memory as a trigger
  const int cnst = 1000000000;
  int *d_ptr;
  hipMalloc(&d_ptr, cnst * sizeof(int));

  dim3 dimBlock( blocksize, 1 );
  dim3 dimGrid( 1, 1 );
  kernelFunc<<<dimGrid, dimBlock>>>();
  hipMemcpy( b, bd, isize, hipMemcpyDeviceToHost ); 
  hipFree( bd );
  hipFree( d_ptr );
  printf(" %i\n", *b);
  return EXIT_SUCCESS;
}
